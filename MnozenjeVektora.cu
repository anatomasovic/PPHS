
#include <hip/hip_runtime.h>
__global__ void matrix_mult (float *result, float *mat1, float *mat2)
{
  const int idx = threadIdx.y * blockDim.x + threadIdx.x;

  float sum_product = 0;
  for (int k = 0; k < 20; k++)
    {
      sum_product += mat1[threadIdx.y * blockDim.x + k] * mat2[k * blockDim.x + threadIdx.x];
    }

  result[idx] = sum_product;
}

/*
import pycuda.autoinit
import pycuda.driver as drv
import numpy as np
from pycuda.compiler import SourceModule

mod = SourceModule(open("mnozenje_vektora.cu").read())

matrix_mult = mod.get_function("matrix_mult")

a = np.ones((20, 20), dtype=np.float32)
b = np.ones((20, 20), dtype=np.float32)

result_gpu = np.empty_like(a)

matrix_mult(drv.Out(result_gpu), drv.In(a), drv.In(b), block=(20,20,1), grid=(1,1))

print (result_gpu)
*/
