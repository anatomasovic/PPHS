
#include <hip/hip_runtime.h>
__global__ void zbroji_matrice (float *dest, float *a, float *b)
{
  const int i = threadIdx.y * blockDim.x + threadIdx.x; //Velicina i pozicija indeksa definirina je pomocu y indeksa niti!
  dest[i] = a[i] + b[i];
}
