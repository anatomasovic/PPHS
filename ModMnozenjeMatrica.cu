#include "hip/hip_runtime.h"
‪#‎include‬ <math.h>
__device__ float rez(float *a, float *b, int i)
{
return 2 * sin(a[i]) + 3 * cos(b[i]); //Pmocna funkcija koja vraca rezultat
}
__global__ void zbrajanje(float *dest, float *a, float *b)
{
const int i = threadIdx.y * blockDim.x + threadIdx.x;
dest[i] = rez(a, b, i);
}

/*
import pycuda.autoinit
import pycuda.driver as drv
import numpy as np
from pycuda.compiler import SourceModule

mod = SourceModule(open("modificirano_mnozenje_matrica.cu").read())
zbrajanje = mod.get_function("zbrajanje")

a = np.ones((10, 10), dtype=np.float32)
b = np.ones((10, 10), dtype=np.float32)

result_gpu = np.empty_like(a)

zbrajanje(drv.Out(result_gpu), drv.In(a), drv.In(b), block = (10, 10, 1), grid = (1, 1))

print (result_gpu)
*/
